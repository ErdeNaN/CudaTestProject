#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

__global__ void addArrays(int* a, int* b, int* c) {
	
	int idx = threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

int main()
{
	int h_a[] = { 10, 20, 30, 40, 50 };
	int h_b[] = { 9, 8, 7, 6, 5 };
	int h_c[5];

	int size = sizeof(int) * 5;
	int *da, *db, *dc;
	hipMalloc((void**)&da, size);
	hipMalloc((void**)&db, size);
	hipMalloc((void**)&dc, size);
	
	hipMemcpy(da, h_a, size, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(db, h_b, size, hipMemcpyKind::hipMemcpyHostToDevice);

	addArrays <<<1, 5 >>> (da, db, dc);

	hipMemcpy(h_c, dc, size, hipMemcpyKind::hipMemcpyDeviceToHost);

	
	for (int i = 0; i < 5; ++i) {
		std::cout << h_c[i] << std::endl;
	}

	return 0;
}
